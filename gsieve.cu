#include "hip/hip_runtime.h"
#include <fstream>
#include <sstream>
#include <regex>
#include <vector>
#include <cassert>

#include "gsieve.cuh"
#include "cub_wrapper.cuh"

#define DEBUG

#include <cub/util_debug.cuh>

pair<Point, Norm> FromVector(const frowvec& v)
{
   Point p;
   auto t = sum(v) / P;
   p[0] = -t;
   for (int i = 0; i < N; ++i)
      p[i + 1] = v[i] - t;
   for (int i = P; i < Pitch; ++i)
      p[i] = 0;

   p.minimize();
   return {p, p.norm()};
}

frowvec ToVector(const Point& p)
{
   frowvec v(N);
   for (int i = 0; i < N; ++i)
      v[i] = p[i + 1] - p[0];
   return v;
}

pair<Point, Norm> Rectify(const Point& p)
{
   // return {p, p.norm()};
   return FromVector(round(ToVector(p)));
}

void List::InitHost(int size)
{
   CubDebugExit(hipHostMalloc(&points, sizeof(Point) * size));
   CubDebugExit(hipHostMalloc(&norms, sizeof(Norm) * size));
}

void List::InitGPU(int size, int gpu)
{
   _gpu = gpu;
   hipSetDevice(gpu);
   CubDebugExit(hipMalloc(&points, sizeof(Point) * size));
   CubDebugExit(hipMalloc(&norms, sizeof(Norm) * size));
}

void List::CopyFromAsync(const List& that, int size, hipStream_t stream, int offset1, int offset2)
{
   CubDebugExit(hipMemcpyAsync(points + offset1, that.points + offset2, sizeof(Point) * size, hipMemcpyDefault, stream));
   CubDebugExit(hipMemcpyAsync(norms + offset1, that.norms + offset2, sizeof(Norm) * size, hipMemcpyDefault, stream));
}

void List::CopyFrom(const List& that, int size, int offset1, int offset2)
{
   CubDebugExit(hipMemcpy(points + offset1, that.points + offset2, sizeof(Point) * size, hipMemcpyDefault));
   CubDebugExit(hipMemcpy(norms + offset1, that.norms + offset2, sizeof(Norm) * size, hipMemcpyDefault));
}

void List::Print(int size, string header)
{
   // TODO: Clean up
   if (_gpu != -1)
   {
      Point ps[size];
      Norm ns[size];

      CubDebugExit(hipSetDevice(_gpu));
      CubDebugExit(hipMemcpy(ps, points, size * sizeof(Point), hipMemcpyDefault));
      CubDebugExit(hipMemcpy(ns, norms, size * sizeof(Norm), hipMemcpyDefault));

      cout << header << ":\n";
      for (int i = 0; i < size; ++i)
      {
         auto v = ToVector(ps[i]);
         printf("%d (%.3f)\t", i, ns[i]);
         v.head(16).raw_print();
      }
   }
   else
   {
      cout << header << ":\n";
      for (int i = 0; i < size; ++i)
      {
         auto v = ToVector(points[i]);
         printf("%d (%.3f)\t", i, norms[i]);
         v.head(16).raw_print();
      }
   }
}

List::~List()
{
   if (_gpu != -1)
   {
      CubDebugExit(hipSetDevice(_gpu));
      CubDebugExit(hipFree(points));
      CubDebugExit(hipFree(norms));
   }
   else
   {
      CubDebugExit(hipHostFree(points));
      CubDebugExit(hipHostFree(norms));
   }
}

GSieve::GSieve(string basis, istream& sample_stream)
   : _sample_stream(sample_stream)
{
   ReadBasis(basis);

   S.InitHost(100000);
   for (int i = 0; i < NGPUS; ++i)
   {
      hostQ[i].InitHost(NumSamples);

      Q[i].InitGPU(NumSamples + 65536, i);
      Q2[i].InitGPU(NumSamples + 65536, i);
      L[i].InitGPU(3500000, i);
      L2[i].InitGPU(3500000, i);

      CubDebugExit(hipSetDevice(i));
      CubDebugExit(hipStreamCreate(&streams[i]));
   }

   cubInit(streams);

   // Prepare S
   best_norm = 1e100;
   for (int i = 0; i < N; ++i)
   {
      tie(S.points[i], S.norms[i]) = FromVector(_B.row(i));
      auto norm = dot(_B.row(i), _B.row(i));
      if (norm < best_norm)
      {
         best_norm = norm;
         shortest_vec = _B.row(i);
      }
   }
   Ssize = N;
}

GSieve::~GSieve()
{
   for (int i = 0; i < NGPUS; ++i)
   {
      CubDebugExit(hipSetDevice(i));
      CubDebugExit(hipStreamDestroy(streams[i]));
   }
}

void GSieve::ReadBasis(string filename)
{
   ifstream fin(filename);
   assert(fin);

   stringstream ss;
   ss << fin.rdbuf();
   auto str = ss.str();

   regex pat{"-?\\d+"}; // Extract all numbers
   sregex_token_iterator p(begin(str), end(str), pat);

   vector<float> nums;
   transform(p, {}, back_inserter(nums), [](string s) { return stof(s); } );
   assert(nums.size() == N * N);

   _B = reshape(fmat(nums), N, N).t();
}

void GSieve::Start()
{
   CubDebugExit(hipSetDevice(0));

   Point* points;
   Norm*  norms;
   CubDebugExit(hipHostMalloc(&points, sizeof(Point) * NumSamples));
   CubDebugExit(hipHostMalloc(&norms, sizeof(Norm) * NumSamples));

   int *new_Lsize;
   CubDebugExit(hipHostMalloc(&new_Lsize, sizeof(int) * NGPUS));

   for (int iterations = 0; iterations < 3000; ++iterations)
   {
      cout << "====== Iteration " << iterations << " ======" << endl;

      CubDebugExit(hipSetDevice(0));
      GenerateSamples(); // Current WRONG

      // Copy GPU 0 samples to CPU to distribute later in Step 0
      CubDebugExit(hipSetDevice(0));
      CubDebugExit(hipMemcpy(points, Q[0].points, sizeof(Point) * NumSamples, hipMemcpyDefault));
      CubDebugExit(hipMemcpy(norms, Q[0].norms, sizeof(Norm) * NumSamples, hipMemcpyDefault));

      // { // REF
      //    List L1, L2;
      //    L1.InitHost(NumSamples);
      //    L2.InitHost(NumSamples);

      //    CubDebugExit(hipMemcpy(L1.points, Q[0].points, sizeof(Point) * NumSamples, hipMemcpyDefault));
      //    CubDebugExit(hipMemcpy(L1.norms, Q[0].norms, sizeof(Norm) * NumSamples, hipMemcpyDefault));

      //    CubDebugExit(hipMemcpy(L2.points, Q[0].points, sizeof(Point) * NumSamples, hipMemcpyDefault));
      //    CubDebugExit(hipMemcpy(L2.norms, Q[0].norms, sizeof(Norm) * NumSamples, hipMemcpyDefault));

      //    GoldenReduce(L1.points, L1.norms, NumSamples, L2.points, L2.norms, NumSamples);

      //    L1.Print(100, "L1");
      // }

      // {
      //    CubDebugExit(hipSetDevice(0));
      //    CubDebugExit(hipMemcpy(L[0].points, Q[0].points + 300, sizeof(Point) * 300, hipMemcpyDefault));
      //    CubDebugExit(hipMemcpy(L[0].norms, Q[0].norms + 300, sizeof(Norm) * 300, hipMemcpyDefault));
      //    Lsize[0] = 300;

      //    CubDebugExit(hipSetDevice(1));
      //    CubDebugExit(hipMemcpy(L[1].points, Q[0].points + 600, sizeof(Point) * 300, hipMemcpyDefault));
      //    CubDebugExit(hipMemcpy(L[1].norms, Q[0].norms + 600, sizeof(Norm) * 300, hipMemcpyDefault));
      //    Lsize[1] = 300;

      //    CubDebugExit(hipSetDevice(2));
      //    CubDebugExit(hipMemcpy(L[2].points, Q[0].points + 900, sizeof(Point) * 300, hipMemcpyDefault));
      //    CubDebugExit(hipMemcpy(L[2].norms, Q[0].norms + 900, sizeof(Norm) * 300, hipMemcpyDefault));
      //    Lsize[2] = 300;
      // }


      for (int i = 0; i < NGPUS; ++i)
      {
         CubDebugExit(hipSetDevice(i));

         // Distribute
         if (i != 0)
         {
            CubDebugExit(hipMemcpyAsync(Q[i].points, points, sizeof(Point) * NumSamples, hipMemcpyDefault, streams[i]));
            CubDebugExit(hipMemcpyAsync(Q[i].norms, norms, sizeof(Norm) * NumSamples, hipMemcpyDefault, streams[i]));
         }
         CubDebugExit(hipMemsetAsync(L[i].points + Lsize[i], 0, 1024 * N, streams[i]));
         CubDebugExit(hipMemsetAsync(L[i].norms + Lsize[i], 0, 1024, streams[i]));
         CubDebugExit(hipMemsetAsync(Q[i].norms + NumSamples, 0, 1024, streams[i]));
         CubDebugExit(hipMemsetAsync(Q2[i].norms + NumSamples, 0, 1024, streams[i]));

         // Q[i].Print(NumSamples, "Q-before");

         reduce<0><<<GridDim, BlockDim, 0, streams[i]>>>(Q[i].points, Q[i].norms, NumSamples, L[i].points, L[i].norms, Lsize[i]);

         // Q[i].Print(NumSamples, "Q");

         Q2[i].CopyFromAsync(Q[i], NumSamples, streams[i]);
         reduce<1><<<GridDim, BlockDim, 0, streams[i]>>>(Q2[i].points, Q2[i].norms, NumSamples, Q[i].points, Q[i].norms, NumSamples);

         // Q2[i].Print(NumSamples, "Q2");

         reduce<2><<<GridDim, BlockDim, 0, streams[i]>>>(L[i].points, L[i].norms, Lsize[i], Q2[i].points, Q2[i].norms, NumSamples);

         // L[i].Print(Lsize[i], "L");

         // Partition 似乎可以輸入輸出相同
         TransformInputIterator<bool, NotReduced, Norm*> itr1(L[i].norms, NotReduced());
         PartitionAsync(L[i].points, itr1, L2[i].points, Lsize[i], i);
         SelectIfAsync(L[i].norms, L[i].norms, Lsize[i], NotReduced(), i);
         GetSelectedSizeAsync(&new_Lsize[i], i);

         hostQ[i].CopyFromAsync(Q2[i], NumSamples, streams[i]);
      }

      for (int i = 0; i < NGPUS; ++i)
      {
         CubDebugExit(hipSetDevice(i));
         CubDebugExit(hipStreamSynchronize(streams[i]));
      }

      // Put reduced vectors (including collisions) onto stack
      for (int i = 0; i < NGPUS; ++i)
      {
         CubDebugExit(hipSetDevice(i));

         int amount = Lsize[i] - new_Lsize[i];
         // S.CopyFrom(L[i], amount, Ssize, new_Lsize[i]);
         CubDebugExit(hipMemcpy(L[i].points, L2[i].points, sizeof(Point) * new_Lsize[i], hipMemcpyDefault));
         CubDebugExit(hipMemcpy(S.points + Ssize, L2[i].points + new_Lsize[i], sizeof(Point) * amount, hipMemcpyDefault));
         CubDebugExit(hipMemcpy(S.norms + Ssize, L[i].norms + new_Lsize[i], sizeof(Norm) * amount, hipMemcpyDefault));

         // Recalculate norm
         for (int k = 0; k < amount; ++k)
           S.norms[Ssize + k] = S.points[Ssize + k].norm();

         Ssize += amount;
         Lsize[i] = new_Lsize[i];
      }
      cout << "NLS: ";
      for (int i = 0; i < NGPUS; ++i)
         cout << Lsize[i] << ' ';
      cout << endl;

      int cnt_r = 0, cnt_nr = 0;
      for (int i = 0; i < NumSamples; ++i)
      {
         // Not reduced -> collect and add to one list
         if (all_of(hostQ, hostQ + NGPUS, [=](const List& l) { return NotReduced()(l.norms[i]); }))
         {
            points[cnt_nr] = hostQ[0].points[i];
            norms[cnt_nr] = hostQ[0].norms[i];
            ++cnt_nr;
         }
         else // Reduced -> throw away collisions, add min rep to stack
         {
            Norm real_norms[NGPUS] {};
            for (int j = 0; j < NGPUS; ++j)
            {
               hostQ[j].points[i].minimize();
               hostQ[j].norms[i] = hostQ[j].points[i].norm(); // Some are -1

               real_norms[j] = hostQ[j].norms[i] + P * hostQ[j].points[i][0] * hostQ[j].points[i][0];
            }

            auto itmin = min_element(real_norms, real_norms + NGPUS);
            int argmin = itmin - real_norms;
            if (NotCollision()(*itmin))
            {
               tie(S.points[Ssize], S.norms[Ssize]) = Rectify(hostQ[argmin].points[i]);

               if (*itmin < best_norm)
               {
                  best_norm = *itmin;
                  shortest_vec = ToVector(S.points[Ssize]);

                  // found_time = system_clock::now();
               }
               ++Ssize;
               ++cnt_r;
            }
         }
      }
      cout << "NR: " << cnt_nr << "  R: " << cnt_r << "  C: " << NumSamples - cnt_nr - cnt_r << endl;
      cout << "S:" << Ssize << endl;

      int min_L = min_element(Lsize, Lsize + NGPUS) - Lsize;
      cout << "Append to List " << min_L << endl;
      CubDebugExit(hipSetDevice(min_L));
      CubDebugExit(hipMemcpy(L[min_L].points + Lsize[min_L], points, sizeof(Point) * cnt_nr, hipMemcpyDefault));
      CubDebugExit(hipMemcpy(L[min_L].norms + Lsize[min_L], norms, sizeof(Norm) * cnt_nr, hipMemcpyDefault));
      Lsize[min_L] += cnt_nr;

      printf("Min Norm = %.3f\n", best_norm);
      cout << '[';
      for (int i = 0; i < N; ++i)
         cout << shortest_vec[i] << ' ';
      cout << ']' << endl;

      // for (int i = 0; i < NGPUS; ++i)
         // L[i].Print(Lsize[i], "L" + to_string(i));
      // S.Print(Ssize, "S");
   }
}

void GSieve::GenerateSamples()
{
   Point points[NumSamples];
   Norm  norms[NumSamples];

   int amount = ::min(NumSamples, Ssize);
   Q[0].CopyFrom(S, amount, 0, Ssize - amount);
   Ssize -= amount;

   for (int i = amount; i < NumSamples; ++i)
   {
      frowvec v(N);
      for (int j = 0; j < N; ++j)
         _sample_stream >> v[j];

      tie(points[i], norms[i]) = FromVector(v);

      float skip;
      _sample_stream >> skip;
   }

   CubDebugExit(hipMemcpy(Q[0].points + amount, points + amount, sizeof(Point) * (NumSamples - amount), hipMemcpyDefault));
   CubDebugExit(hipMemcpy(Q[0].norms + amount, norms + amount, sizeof(Norm) * (NumSamples - amount), hipMemcpyDefault));
}

void GSieve::GoldenReduce(Point* gs, Norm* gns, size_t gsize, const Point* hs, const Norm* hns, size_t hsize)
{
   for (int i = 0; i < gsize; ++i)
   {
      Point& g = gs[i];
      Norm&  gg = gns[i];

      float min_norm = gg + P * g[0] * g[0];
      // cout << min_norm << endl;
      for (int j = 0; j < hsize; ++j)
      {
         const Point& h = hs[j];
         const Norm  hh = hns[j];

         for (int rot = 0; rot < P; ++rot)
         {
            float gh {};
            for (int k = 0; k < P; ++k)
               gh += g[k] * h[(k + rot) % P];

            float best_m = 0;

            for (int k = 0; k < P; ++k)
            {
               float p = g[k], q = h[(k + rot) % P];

               float uu = gg + P * p * p,
                     uv = gh + P * p * q,
                     vv = hh + P * q * q;

               float m = std::round(uv / uu);
               float new_norm = uu - 2 * m * uv + m * m * vv;

               if (new_norm < min_norm && m != 0)
               {
                  // printf("%d, %d, %d (%d) -> %.0f, %.0f, %.0f -> %.0f (%.0f, %.0f)\n",
                  //    i, j, k, rot, uu, uv, vv, m, new_norm, min_norm);
                  min_norm = new_norm;
                  best_m = m;
               }
            }

            for (int k = 0; k < P; ++k)
               g[k] -= best_m * h[(k + rot) % P];

            gg += best_m * best_m * hh - best_m * 2 * gh;
         }
      }
   }
}
