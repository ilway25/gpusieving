#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>

#include "kernels.cuh"

__device__ float tmp;

const int NumPrefetch = CUB_QUOTIENT_FLOOR(4 * BlockDim, Pitch);

using shared_t = union
{
   float block[BlockDim][4];
   float sep[NumPrefetch][RakeWidth][NT];
   float linear[NumPrefetch][Pitch];
};

__global__
void reduce(Point* gs, Norm* gns, size_t g_size, const Point* hs, const Norm* hns, size_t h_size)
{
   const int subidx  = threadIdx.x % RakeWidth;
   const int subinst = threadIdx.x / RakeWidth;

   float* g_ptr = reinterpret_cast<float*>(gs);
   const float* h_ptr = reinterpret_cast<const float*>(hs);

   hipcub::CacheModifiedInputIterator<hipcub::LOAD_LDG, float> g_in(g_ptr);
   hipcub::CacheModifiedInputIterator<hipcub::LOAD_LDG, float> h_in(h_ptr);

   using BlockLoadT = hipcub::BlockLoad<decltype(g_in), BlockDim, NT, hipcub::BLOCK_LOAD_VECTORIZE>;
   using BlockStoreT = hipcub::BlockStore<float*, BlockDim, NT, hipcub::BLOCK_STORE_VECTORIZE>;
   using BlockLoadVT = hipcub::BlockLoad<decltype(h_in), BlockDim, 4, hipcub::BLOCK_LOAD_VECTORIZE>;

   union {
      typename BlockLoadT::TempStorage load;
      typename BlockStoreT::TempStorage store;
      typename BlockLoadVT::TempStorage loadv;
   } shared;


   for (int g_base = blockIdx.x * InstPerBlock; g_base < g_size; g_base += GridDim * InstPerBlock)
   {
      const int g_idx = g_base + subinst;

      float g[NT], gg;

      BlockLoadT(shared.load).Load(g_in + g_base * Pitch, g);
      gg = gns[g_idx];

      float min_norm = gg + P * (g_in + g_base * Pitch)[0] * (g_in + g_base * Pitch)[0];

      __shared__ alignas(128) shared_t prefetch;
      __shared__ float prefetch_n[BlockDim];

      for (int h_base = 0; h_base < h_size; h_base += NumPrefetch)
      {
         BlockLoadVT(shared.loadv).Load(h_in + h_base * Pitch, prefetch.block[threadIdx.x]);
         prefetch_n[threadIdx.x] = hns[h_base + threadIdx.x];
         __syncthreads();

         for (int i = 0; i < NumPrefetch && h_base + i < h_size; ++i)
         {
            const int h_idx = h_base + i;

            // g_buf has no zero padding
            using sep = float[RakeWidth][NT];
            __shared__ float h_buf[Pitch * 2];

            if (threadIdx.x < P)
            {
               h_buf[threadIdx.x] = prefetch.linear[i][threadIdx.x];
               h_buf[threadIdx.x + P] = 0;
            }

            if (threadIdx.x < Pitch * 2 - P * 2)
               h_buf[threadIdx.x + 2 * P] = 0;

            float hh = prefetch_n[i];

            __syncthreads();

            for (int rot = 0; rot < P; ++rot)
            {
               float q_best {};
               float gh {};

               float h[NT];
               for (int j = 0; j < NT; ++j)
                  h[j] = (*(sep*)(&h_buf[rot]))[subidx][j];

               for (int j = 0; j < NT; ++j)
                  gh += g[j] * h[j];

               for (int j = 1; j < RakeWidth; j *= 2)
                  gh += __shfl_xor(gh, j);

               for (int j = 0; j < NT; ++j)
               {
                  float uu = gg + P * g[j] * g[j],
                        uv = gh + P * g[j] * h[j],
                        vv = hh + P * h[j] * h[j];

                  float q = rintf(uv / uu);
                  float new_norm = uu - 2 * q * uv + q * q * vv;

                  if (new_norm < min_norm && q != 0 && subidx * NT + j < P)
                  {
                     // printf("%d, %d, %d (%d) -> %.0f, %.0f, %.0f -> %.0f (%.0f, %.0f)\n",
                     //    g_idx, h_idx, subidx * NT + j, rot,
                     //    uu, uv, vv, q, new_norm, min_norm);

                     min_norm = new_norm;
                     q_best = q;
                  }
               }

               for (int j = 1; j < RakeWidth; j *= 2)
               {
                  float min_norm_t = __shfl_xor(min_norm, j);
                  float q_best_t = __shfl_xor(q_best, j);

                  if (min_norm_t < min_norm || min_norm_t == min_norm && (subidx ^ j) >= subidx)
                  {
                     min_norm = min_norm_t;
                     q_best = q_best_t;
                  }
               }

               for (int j = 0; j < NT; ++j)
                  g[j] -= q_best * h[j];
               gg += q_best * q_best * hh - q_best * 2 * gh;

               // if (q_best != 0 && subidx == 0)
               //    printf("%d %d %d %f\n", g_idx, h_idx, rot, q_best);

               if (threadIdx.x == 0)
                  h_buf[P + rot] = h[0];

               __syncthreads();
            }
         }
      }

      BlockStoreT(shared.store).Store(g_ptr + g_base * Pitch, g);
      gns[g_idx] = gg;
   }
}
